#include "hip/hip_runtime.h"
#include "A.h"
#include <hipcub/hipcub.hpp>
#include <limits.h>
#include <time.h>

__global__ void calcular_distancia(Point* all_points, char* all_labels, float* all_distances, char* label_distances, int total_points, int k, Point to_evaluate){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < total_points){
        all_distances[i] = ((to_evaluate.x - all_points[i].x) * ((to_evaluate.x - all_points[i].x))) + ((to_evaluate.y - all_points[i].y) * (to_evaluate.y - all_points[i].y));
        label_distances[i] = all_labels[i];
    }
}

int unindo_groups(Point** all_points, char** all_labels, Group* groups, int n_groups){
    int total_points = 0;
    for(int i = 0; i < n_groups; i++) total_points += groups[i].length;

    *all_points = (Point*)malloc(sizeof(Point) * total_points);
    *all_labels = (char*)malloc(sizeof(char) * total_points);

    int indice = 0;
    for(int i = 0; i < n_groups; i++){
        for(int j = 0; j < groups[i].length; j++){
            (*all_points)[indice] = groups[i].points[j];
            (*all_labels)[indice] = groups[i].label;
            indice++;
        }
    }
    return total_points;
}

void ordenar_distancias(float* all_distances, float* sorted_distances, char* all_labels, char* sorted_labels, int total_distances){
    void* aux = NULL;
    size_t aux_bytes = 0;
    
    hipcub::DeviceRadixSort::SortPairs(aux, aux_bytes, all_distances, sorted_distances, all_labels, sorted_labels, total_distances);

    hipMalloc(&aux, aux_bytes);

    hipcub::DeviceRadixSort::SortPairs(aux, aux_bytes, all_distances, sorted_distances, all_labels, sorted_labels, total_distances);
    hipFree(aux);

}

char calcular_mais_frequente(char* labels, int k){
    int freq[256] = {0};
    for(int i = 0; i < k; i++){
        unsigned char indice = (unsigned char)labels[i];
        freq[indice]++;
    }

    char mais_frequente = 0;
    int max_freq = INT_MIN;
    for(int i = 0; i <256; i++){
        if(freq[i] > max_freq){
            max_freq = freq[i];
            mais_frequente = char(i);
        }
    }
    return mais_frequente;
}

char knn(Point* all_points, char* all_labels, int total_points, int k, Point to_evaluate){
    float *dev_all_distances;
    char *dev_label_distances;
    Point *dev_all_points;
    char *dev_all_labels;
    
    hipMalloc((void**)&dev_all_points, total_points * sizeof(Point));

    hipMalloc((void**)&dev_all_labels, total_points * sizeof(char));

    hipMalloc((void**)&dev_all_distances, total_points * sizeof(float));

    hipMalloc((void**)&dev_label_distances, total_points * sizeof(char));

    hipMemcpy(dev_all_points, all_points, total_points * sizeof(Point), hipMemcpyHostToDevice);

    hipMemcpy(dev_all_labels, all_labels, total_points * sizeof(char), hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (total_points + (threads - 1)) / threads;

    calcular_distancia<<<blocks, threads>>>(dev_all_points, dev_all_labels, dev_all_distances, dev_label_distances, total_points, k, to_evaluate);
    hipDeviceSynchronize();

    float* dev_sorted_distances;
    char* dev_sorted_labels;

    hipMalloc(&dev_sorted_distances, total_points * sizeof(float));

    hipMalloc(&dev_sorted_labels, total_points * sizeof(char));

    ordenar_distancias(dev_all_distances, dev_sorted_distances, dev_label_distances, dev_sorted_labels, total_points);

    char* k_menores_labels = (char*)malloc(k * sizeof(char));
    hipMemcpy(k_menores_labels, dev_sorted_labels, k * sizeof(char), hipMemcpyDeviceToHost);

    char resultado = calcular_mais_frequente(k_menores_labels, k);
    return resultado;
}

int main(){
    int n_groups = parse_number_of_groups();
    
    Group *groups = (Group *) malloc(sizeof(Group) * n_groups);
    
    for (int i = 0; i < n_groups; i++) {
        groups[i] = parse_next_group();
    }
    
    
    int k = parse_k();
    Point to_evaluate = parse_point();
    
    Point *all_points;
    char *all_labels;
    int total_points = unindo_groups(&all_points, &all_labels, groups, n_groups);
    printf("%c", knn(all_points, all_labels, total_points, k, to_evaluate));
}